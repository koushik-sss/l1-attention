#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
<<<<<<< HEAD
=======
#include <iostream>
>>>>>>> 11dc1eb (new files)

template <typename scalar_t>
__device__  __forceinline__ scalar_t sign(scalar_t x)
{ 
<<<<<<< HEAD
	scalar_t t = x < 0 ? -1 : 0;
	return x > 0 ? 1 : t;
=======
    scalar_t t = x < 0 ? -1 : 0;
    return x > 0 ? 1 : t;
>>>>>>> 11dc1eb (new files)
}

template <typename scalar_t>
__device__  __forceinline__ void fastAtomicAdd2(
<<<<<<< HEAD
	torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> out, 
	int i0, int i1, int i2, int i3, scalar_t v)
{
	// convenience wrapper function around
	// fastAtomicAdd for 4-D tensors. 
	int index = i0*out.stride(0) + i1*out.stride(1) + i2*out.stride(2) + i3*out.stride(3);
	at::native::fastAtomicAdd(out.data(), index, 1, v, true); 
=======
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> out, 
    int i0, int i1, int i2, int i3, scalar_t v)
{
    // convenience wrapper function around
    // fastAtomicAdd for 4-D tensors.
    int index = i0*out.stride(0) + i1*out.stride(1) + i2*out.stride(2) + i3*out.stride(3);
    at::native::fastAtomicAdd(out.data(), index, 1, v, true); 
>>>>>>> 11dc1eb (new files)
}

template <typename scalar_t>
__global__ void l1attn_cuda_forward_kernelX(
<<<<<<< HEAD
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		attn,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width)
{
	__shared__ scalar_t acc[32];
	
	int tix = threadIdx.x; // [0 .. 31]. 
	// tix operates within across the width dimension (reduction dim) 
	int h = blockIdx.x % n_heads; 
	int t = blockIdx.x / n_heads; 
	int s = blockIdx.y; 
	int b = blockIdx.z; 
	
	int width32 = (width + 31) / 32; 
	scalar_t f = 0.0; 
	for(int w = 0; w < width32; w++) { 
		int o = w*32+tix; 
		if(o < width)
			f += abs(q[b][t][h][o] - k[b][s][h][o]); 
	}
	acc[tix] = f * scale; 
	if(tix < 16) { 
		acc[tix] += acc[tix + 16];
		__syncthreads(); // why is this needed ??? 
		acc[tix] += acc[tix + 8 ];
		__syncthreads(); // threads in a warp should be synchronous.
		acc[tix] += acc[tix + 4 ];
		__syncthreads(); // experiment: it's totally needed! 
		acc[tix] += acc[tix + 2 ];
		__syncthreads();
		acc[tix] += acc[tix + 1 ];
		__syncthreads();
		if(tix == 0){
			attn[b][s][t][h] = acc[tix]; 
		}
	}
}

#define	BLKSIZ 16
template <typename scalar_t>
__global__ void l1attn_cuda_forward_kernel16(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		attn,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width)
{
	// q and k must be bhtw and bhsw respectively
	// despite the name of this function, it only operates on 
	// width 32 q and k tensors, in blocks of 16 x 16
	// Larger would require more per-warp memory or use of registers: 
	// 2 x 16 x 32 x 4 bytes = 4096 kB per block, so each SM can have 12 blocks. 
	
	int w = threadIdx.x; // t thread [0 .. 15]. 
	int u = threadIdx.y; // t for q, s for k,  [0 .. 15]. 
	int tb = blockIdx.x; // t block
	int sb = blockIdx.y; // s block
	int h = blockIdx.z % n_heads; // head
	int b = blockIdx.z / n_heads; // block
	
	// each block computes a BLKSIZ x BLKSIZ block of the attention matrix
	// a block is 256 threads
	// so, each thread loads one value from each q,k
	__shared__ scalar_t qc[BLKSIZ][32]; // q cache 
	__shared__ scalar_t kc[BLKSIZ][32]; // k cache
	
	//reshape to 8 warps, 32 threads - better mem throughput
	int tid = u*BLKSIZ + w; 
	int cw = tid % 32; // cache w
	int cu = tid / 32; // cache u
	int t = tb * BLKSIZ + cu; 
	int s = sb * BLKSIZ + cu; 
	
	qc[cu  ][cw] = q[b][h][t][cw]; // each thread reads/writes one fp32
	qc[cu+8][cw] = q[b][h][t+8][cw];
	kc[cu  ][cw] = k[b][h][s][cw];
	kc[cu+8][cw] = k[b][h][s+8][cw];
	
	__syncthreads();
	
	// simple approach: each thread computes one attention value
	// redefine t and s
	t = u; // so q is shared between threads in the same warp
	s = w; 
	scalar_t f = 0.0; 
	for(int o=0; o < 32; o++){
		f += abs(qc[t][o] - kc[s][o]); // ultimately want these to be registers
	}
	// back to global
	t = tb * BLKSIZ + u; 
	s = sb * BLKSIZ + w; 
	attn[b][s][t][h] = f * scale; // this is unaligned. ought to fix.
}

// template <typename scalar_t>
// __global__ void l1attn_cuda_backward_kernel_old(
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_attn,
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		q,
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		k,
// 		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_q,
// 		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_k,
// 		const scalar_t scale, 
// 		const int bs, const int n_ctx, const int n_heads, const int width ) 
// {
// 	// reduction (across s and t) has to be done within a thread warp: 
// 	// can't have different warps write the same memory. 
// 	// they will interfere / not give the correct answer!
// 	
// 	int indx = threadIdx.x + blockIdx.x * blockDim.x; // 1D
// 	
// 	if(indx < bs*n_ctx*n_ctx*n_heads){
// 		// again, output indexing b/c thread blocks can't overlap writes.
// 		// see note in forward kernel.
// 		int j = indx; 
// 		int h = j % n_heads; 
// 		j /= n_heads; 
// 		int s = j % n_ctx; 
// 		j /= n_ctx; 
// 		int t = j % n_ctx; 
// 		j /= n_ctx; 
// 		int b = j % bs; 
// 		
// 		scalar_t d_a = d_attn[b][s][t][h]; 
// 		for(int w = 0; w < width; w++){
// 			scalar_t ws = q[b][t][h][w] - k[b][s][h][w];
// 			ws = sign(ws) * scale; 
// 			// atomicAdd((scalar_t*)&(d_q[b][t][h][w]), ws * d_a);
// 			// atomicAdd((scalar_t*)&(d_k[b][s][h][w]), -1*ws * d_a);
// 			fastAtomicAdd2(d_q, b,t,h,w, ws * d_a);
// 			fastAtomicAdd2(d_k, b,s,h,w, -1*ws * d_a);
// 		}
// 	}
// } 

template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attnq,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attnk,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_q,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_k,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width ) 
{
	__shared__ scalar_t acc_dq[32];
	__shared__ scalar_t acc_dk[32];
	
	int tix = threadIdx.x; // [0 .. 31].
	int h = blockIdx.x % n_heads; 
	int r = blockIdx.x / n_heads; // r is t for q, s for k.
	int w = blockIdx.y; 
	int b = blockIdx.z; 
		
	int ctx32 = (n_ctx + 31) / 32; 
	scalar_t dq = 0.0; 
	scalar_t dk = 0.0; 
	
	scalar_t qq = q[b][w][h][r]; 
	for(int o = 0; o < ctx32; o++) { 
		int s = o*32+tix; 
		if(s < n_ctx){ 
			// all this would work better if n_ctx were a multiple of 32. 
			scalar_t ws = qq - k[b][w][h][s];
			ws = sign(ws) * scale; 
			scalar_t d_a = d_attnq[b][r][h][s]; 
			dq += ws * d_a; 
		}
	}
	
	scalar_t kk = k[b][w][h][r]; 
	for(int o = 0; o < ctx32; o++) { 
		int t = o*32+tix; 
		if(t < n_ctx){
			scalar_t ws = q[b][w][h][t] - kk;
			ws = sign(ws) * scale; 
			scalar_t d_a = d_attnk[b][r][h][t]; 
			dk -= ws * d_a; 
		}
	}
	
	acc_dq[tix] = dq;
	acc_dk[tix] = dk;
	if(tix < 16) { 
		acc_dq[tix] += acc_dq[tix + 16];
		acc_dk[tix] += acc_dk[tix + 16];
		__syncthreads(); 
		acc_dq[tix] += acc_dq[tix + 8 ];
		acc_dk[tix] += acc_dk[tix + 8 ];
		__syncthreads(); 
		acc_dq[tix] += acc_dq[tix + 4 ];
		acc_dk[tix] += acc_dk[tix + 4 ];
		__syncthreads();
		acc_dq[tix] += acc_dq[tix + 2 ];
		acc_dk[tix] += acc_dk[tix + 2 ];
		__syncthreads();
		acc_dq[tix] += acc_dq[tix + 1 ];
		acc_dk[tix] += acc_dk[tix + 1 ];
		__syncthreads();
		if(tix == 0){
			d_q[b][r][h][w] = acc_dq[tix];
			d_k[b][r][h][w] = acc_dk[tix]; 
		}
	}
}

template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel16(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attn,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_q,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_k,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width) 
{
	// q and k must be bhtw and bhsw respectively
	// d_attn must be bhts (usually bsth)
	// output is bhtzw / bhszw, where z is an extra reduction dim over 16x16 
	
	int v = threadIdx.x; // thread [0 .. 15]. 
	int r = threadIdx.y; // t for q, s for k,  [0 .. 15]. 
	int sb = blockIdx.x; // s block
	int tb = blockIdx.y; // t block
	int h = blockIdx.z % n_heads; // head
	int b = blockIdx.z / n_heads; // block
	
	// each block computes a BLKSIZ x 32 block of d_q, d_k
	// a block is 256 threads
	// so, each thread loads four values from each q,k
	// and one from d_attn
	__shared__ scalar_t dac[BLKSIZ][BLKSIZ]; // d_attn cache 
	__shared__ scalar_t qc[BLKSIZ][32]; // q cache 
	__shared__ scalar_t kc[BLKSIZ][32]; // k cache
	
	// this will be partly uncoalesced. 
	int s = sb * BLKSIZ + v; 
	int t = tb * BLKSIZ + r; 
	dac[r][v] = d_attn[b][h][t][s]; 
	
	int tid = r*BLKSIZ + v; 
	int cw = tid % 32; // cache w
	int cr = tid / 32; // cache r
	s = sb * BLKSIZ + cr; 
	t = tb * BLKSIZ + cr; 
	
	qc[cr  ][cw] = q[b][h][t][cw]; // each thread reads one fp32
	qc[cr+8][cw] = q[b][h][t+8][cw];
	kc[cr  ][cw] = k[b][h][s][cw]; // full 32-wide load
	kc[cr+8][cw] = k[b][h][s+8][cw];
	__syncthreads();
	
	scalar_t dq, dk, qq, kk;
	for(int p = 0; p < 32; p += 16){
		int w = v + p; 
		dq = 0.0;
		t = r; 
		qq = qc[t][w]; 
		for(s = 0; s < BLKSIZ; s++){
			scalar_t ws = qq - kc[s][w];
			ws = sign(ws) * scale; 
			dq += ws * dac[t][s]; 
		}
		t = tb * BLKSIZ + r;
		//d_q[b][t][h][z][w] = dq; 
		fastAtomicAdd2( d_q, b,h,t,w, dq ); // ouch. o/w need too much mem.
	
		dk = 0.0; 
		s = r; 
		kk = kc[s][w]; 
		for(t = 0; t < BLKSIZ; t++){
			scalar_t ws = qc[t][w] - kk;
			ws = sign(ws) * scale; 
			dk -= ws * dac[t][s]; 
		}
		s = sb * BLKSIZ + r; 
		//d_k[b][s][h][z][w] = dk; 
		fastAtomicAdd2( d_k, b,h,s,w, dk ); 
	}
}

std::vector<torch::Tensor> l1attn_cuda_forward(
		torch::Tensor q,
		torch::Tensor k) {
  
	int bs = q.sizes()[0]; 
	int n_ctx = q.sizes()[1]; 
	int n_heads = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad()); //better way to do this? 
	
	auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
	
	const dim3 numBlocks(n_heads*n_ctx, n_ctx, bs); // x, y, z
	const dim3 threadsPerBlock(32, 1, 1);
	
	double scale = -1.0 / sqrt(width); 
		
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel", ([&] {
		l1attn_cuda_forward_kernelX<scalar_t><<<numBlocks, threadsPerBlock>>>(
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_forward16(
		torch::Tensor q,
		torch::Tensor k) {
  
	int bs = q.sizes()[0]; 
	int n_heads = q.sizes()[1];
	int n_ctx = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad()); //better way to do this? 
	
	auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
	
	const dim3 numBlocks(n_ctx/BLKSIZ, n_ctx/BLKSIZ, bs*n_heads); // x, y, z
	const dim3 threadsPerBlock(BLKSIZ, BLKSIZ, 1);
	
	double scale = -1.0 / sqrt(width); 
		
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel16", ([&] {
		l1attn_cuda_forward_kernel16<scalar_t><<<numBlocks, threadsPerBlock>>>(
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	// output is bhts; should be bsth to work with everything else.
	// attn = attn.transpose(1,3).contiguous(); 
	
	return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_backward(
		torch::Tensor d_attnq,
		torch::Tensor d_attnk,
		torch::Tensor q,
		torch::Tensor k) 
{
	int bs = q.sizes()[0]; // permuted in python driver!!!
	int width = q.sizes()[1];
	int n_heads = q.sizes()[2]; 
	int n_ctx = q.sizes()[3]; 
	
	double scale = -1.0 / sqrt(width);
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad());
	
	auto d_q = torch::zeros({bs, n_ctx, n_heads, width}, options);
	auto d_k = torch::zeros({bs, n_ctx, n_heads, width}, options);
	
	const dim3 numBlocks(n_heads*n_ctx, width, bs); // x, y, z
	const dim3 threadsPerBlock(32, 1, 1);
	
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel", ([&] {
		l1attn_cuda_backward_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
			d_attnq.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_attnk.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {d_q, d_k};
}

std::vector<torch::Tensor> l1attn_cuda_backward16(
		torch::Tensor d_attn,
		torch::Tensor q,
		torch::Tensor k) 
{
	int bs = q.sizes()[0]; 
	int n_heads = q.sizes()[1]; 
	int n_ctx = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	double scale = -1.0 / sqrt(width);
	int zwidth = n_ctx / 16; 
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad());
	
	auto d_q = torch::zeros({bs, n_heads, n_ctx, width}, options);
	auto d_k = torch::zeros({bs, n_heads, n_ctx, width}, options);
	
	// const dim3 dimBlocks(32, 8); // x, y, z
	const dim3 numBlocks(zwidth, zwidth, n_heads*bs); // x, y, z
	const dim3 threadsPerBlock(16, 16, 1); 
	
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel16", ([&] {
		l1attn_cuda_backward_kernel16<scalar_t><<<numBlocks, threadsPerBlock>>>(
			d_attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	// bhtw -> bthw -- really need to change everything in the lib! 
	d_q = d_q.transpose_(1,2).contiguous();
	d_k = d_k.transpose_(1,2).contiguous(); 
	
	return {d_q, d_k}; // reduce along the zsize dim
=======
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        q,
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        k,
        torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        attn,
        const scalar_t scale, 
        const int bs, const int n_ctx, const int n_heads, const int width)
{
    __shared__ scalar_t acc[32];
    
    int tix = threadIdx.x; // [0 .. 31]. 
    // tix operates within across the width dimension (reduction dim) 
    int h = blockIdx.x % n_heads; 
    int t = blockIdx.x / n_heads; 
    int s = blockIdx.y; 
    int b = blockIdx.z; 
    
    int width32 = (width + 31) / 32; 
    scalar_t f = 0.0; 
    for(int w = 0; w < width32; w++) { 
        int o = w*32+tix; 
        if(o < width)
            f += abs(q[b][t][h][o] - k[b][s][h][o]); 
    }
    acc[tix] = f * scale; 
    if(tix < 16) { 
        acc[tix] += acc[tix + 16];
        __syncthreads(); // why is this needed ??? 
        acc[tix] += acc[tix + 8 ];
        __syncthreads(); // threads in a warp should be synchronous.
        acc[tix] += acc[tix + 4 ];
        __syncthreads(); // experiment: it's totally needed! 
        acc[tix] += acc[tix + 2 ];
        __syncthreads();
        acc[tix] += acc[tix + 1 ];
        __syncthreads();
        if(tix == 0){
            attn[b][s][t][h] = acc[tix]; 
        }
    }
}

#define BLKSIZ 16

#define FORWARD_KERNEL_IMPL(WIDTH) \
template <typename scalar_t> \
__global__ void l1attn_cuda_forward_kernel16_w##WIDTH( \
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> q, \
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> k, \
        torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> attn, \
        const scalar_t scale, \
        const int bs, const int n_ctx, const int n_heads, const int width) \
{ \
    /* q and k must be bhtw and bhsw respectively */ \
    /* despite the name of this function, it only operates on */ \
    /* width 32 q and k tensors, in blocks of 16 x 16 */ \
    /* Larger would require more per-warp memory or use of registers: */ \
    /* 2 x 16 x 32 x 4 bytes = 4096 kB per block, so each SM can have 12 blocks. */ \
    __shared__ scalar_t qc[BLKSIZ][WIDTH]; \
    __shared__ scalar_t kc[BLKSIZ][WIDTH]; \
    \
    int w = threadIdx.x; /* t thread [0 .. 15]. */ \
    int u = threadIdx.y; /* t for q, s for k,  [0 .. 15]. */ \
    int tb = blockIdx.x; /* t block */ \
    int sb = blockIdx.y; /* s block */ \
    int h = blockIdx.z % n_heads; /* head */ \
    int b = blockIdx.z / n_heads; /* block */ \
    \
    /* each block computes a BLKSIZ x BLKSIZ block of the attention matrix */ \
    /* a block is 256 threads */ \
    /* so, each thread loads one value from each q,k */ \
    int tid = u*BLKSIZ + w; \
    int cw = tid % WIDTH; /* cache w */ \
    int cu = tid / WIDTH; /* cache u */ \
    int t = tb * BLKSIZ + cu; \
    int s = sb * BLKSIZ + cu; \
    \
    if (cu < BLKSIZ) { \
        qc[cu][cw] = q[b][h][t][cw]; /* each thread reads/writes one fp32 */ \
        kc[cu][cw] = k[b][h][s][cw]; \
    } \
    \
    __syncthreads(); \
    \
    /* simple approach: each thread computes one attention value */ \
    /* redefine t and s */ \
    t = u; /* so q is shared between threads in the same warp */ \
    s = w; \
    scalar_t f = 0.0; \
    for(int o=0; o < WIDTH; o++){ \
        f += abs(qc[t][o] - kc[s][o]); /* ultimately want these to be registers */ \
    } \
    /* back to global */ \
    t = tb * BLKSIZ + u; \
    s = sb * BLKSIZ + w; \
    attn[b][s][t][h] = f * scale; /* this is unaligned. ought to fix. */ \
}

FORWARD_KERNEL_IMPL(16)
FORWARD_KERNEL_IMPL(32)
FORWARD_KERNEL_IMPL(64)

template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel(
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        d_attnq,
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        d_attnk,
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        q,
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        k,
        torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        d_q,
        torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
        d_k,
        const scalar_t scale, 
        const int bs, const int n_ctx, const int n_heads, const int width ) 
{
    __shared__ scalar_t acc_dq[32];
    __shared__ scalar_t acc_dk[32];
    
    int tix = threadIdx.x; // [0 .. 31].
    int h = blockIdx.x % n_heads; 
    int r = blockIdx.x / n_heads; // r is t for q, s for k.
    int w = blockIdx.y; 
    int b = blockIdx.z; 
        
    int ctx32 = (n_ctx + 31) / 32; 
    scalar_t dq = 0.0; 
    scalar_t dk = 0.0; 
    
    scalar_t qq = q[b][w][h][r]; 
    for(int o = 0; o < ctx32; o++) { 
        int s = o*32+tix; 
        if(s < n_ctx){ 
            // all this would work better if n_ctx were a multiple of 32. 
            scalar_t ws = qq - k[b][w][h][s];
            ws = sign(ws) * scale; 
            scalar_t d_a = d_attnq[b][r][h][s]; 
            dq += ws * d_a; 
        }
    }
    
    scalar_t kk = k[b][w][h][r]; 
    for(int o = 0; o < ctx32; o++) { 
        int t = o*32+tix; 
        if(t < n_ctx){
            scalar_t ws = q[b][w][h][t] - kk;
            ws = sign(ws) * scale; 
            scalar_t d_a = d_attnk[b][r][h][t]; 
            dk -= ws * d_a; 
        }
    }
    
    acc_dq[tix] = dq;
    acc_dk[tix] = dk;
    if(tix < 16) { 
        acc_dq[tix] += acc_dq[tix + 16];
        acc_dk[tix] += acc_dk[tix + 16];
        __syncthreads(); 
        acc_dq[tix] += acc_dq[tix + 8 ];
        acc_dk[tix] += acc_dk[tix + 8 ];
        __syncthreads(); 
        acc_dq[tix] += acc_dq[tix + 4 ];
        acc_dk[tix] += acc_dk[tix + 4 ];
        __syncthreads();
        acc_dq[tix] += acc_dq[tix + 2 ];
        acc_dk[tix] += acc_dk[tix + 2 ];
        __syncthreads();
        acc_dq[tix] += acc_dq[tix + 1 ];
        acc_dk[tix] += acc_dk[tix + 1 ];
        __syncthreads();
        if(tix == 0){
            d_q[b][r][h][w] = acc_dq[tix];
            d_k[b][r][h][w] = acc_dk[tix]; 
        }
    }
}

#define BACKWARD_KERNEL_IMPL(WIDTH) \
template <typename scalar_t> \
__global__ void l1attn_cuda_backward_kernel16_w##WIDTH( \
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_attn, \
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> q, \
        const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> k, \
        torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_q, \
        torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_k, \
        const scalar_t scale, \
        const int bs, const int n_ctx, const int n_heads, const int width) \
{ \
    /* q and k must be bhtw and bhsw respectively */ \
    /* d_attn must be bhts (usually bsth) */ \
    /* output is bhtzw / bhszw, where z is an extra reduction dim over 16x16 */ \
    __shared__ scalar_t dac[BLKSIZ][BLKSIZ]; /* d_attn cache */ \
    __shared__ scalar_t qc[BLKSIZ][WIDTH]; /* q cache */ \
    __shared__ scalar_t kc[BLKSIZ][WIDTH]; /* k cache */ \
    \
    int v = threadIdx.x; /* thread [0 .. 15]. */ \
    int r = threadIdx.y; /* t for q, s for k,  [0 .. 15]. */ \
    int sb = blockIdx.x; /* s block */ \
    int tb = blockIdx.y; /* t block */ \
    int h = blockIdx.z % n_heads; /* head */ \
    int b = blockIdx.z / n_heads; /* block */ \
    \
    /* this will be partly uncoalesced. */ \
    int s = sb * BLKSIZ + v; \
    int t = tb * BLKSIZ + r; \
    dac[r][v] = d_attn[b][h][t][s]; \
    \
    int tid = r*BLKSIZ + v; \
    int cw = tid % WIDTH; /* cache w */ \
    int cr = tid / WIDTH; /* cache r */ \
    s = sb * BLKSIZ + cr; \
    t = tb * BLKSIZ + cr; \
    \
    if (cr < BLKSIZ) { \
        qc[cr][cw] = q[b][h][t][cw]; /* each thread reads one fp32 */ \
        kc[cr][cw] = k[b][h][s][cw]; /* full 32-wide load */ \
    } \
    __syncthreads(); \
    \
    scalar_t dq, dk, qq, kk; \
    for(int p = 0; p < WIDTH; p += BLKSIZ){ \
        int w = v + p; \
        if (w < width) { \
            dq = 0.0; \
            t = r; \
            qq = qc[t][w]; \
            for(s = 0; s < BLKSIZ; s++){ \
                scalar_t ws = qq - kc[s][w]; \
                ws = sign(ws) * scale; \
                dq += ws * dac[t][s]; \
            } \
            t = tb * BLKSIZ + r; \
            fastAtomicAdd2(d_q, b,h,t,w, dq); \
        \
            dk = 0.0; \
            s = r; \
            kk = kc[s][w]; \
            for(t = 0; t < BLKSIZ; t++){ \
                scalar_t ws = qc[t][w] - kk; \
                ws = sign(ws) * scale; \
                dk -= ws * dac[t][s]; \
            } \
            s = sb * BLKSIZ + r; \
            fastAtomicAdd2(d_k, b,h,s,w, dk); \
        } \
    } \
}

BACKWARD_KERNEL_IMPL(16)
BACKWARD_KERNEL_IMPL(32)
BACKWARD_KERNEL_IMPL(64)

std::vector<torch::Tensor> l1attn_cuda_forward(
        torch::Tensor q,
        torch::Tensor k) {
  
    int bs = q.sizes()[0]; 
    int n_ctx = q.sizes()[1]; 
    int n_heads = q.sizes()[2]; 
    int width = q.sizes()[3];
    
    auto options = torch::TensorOptions()
        .dtype(q.dtype())
        .device(q.device())
        .requires_grad(q.requires_grad()); //better way to do this?
    
    auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
    
    const dim3 numBlocks(n_heads*n_ctx, n_ctx, bs); // x, y, z
    const dim3 threadsPerBlock(32, 1, 1);
    
    double scale = -1.0 / sqrt(width); 
        
    AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel", ([&] {
        // std::cout << "CUDA: Executing general forward kernel" << std::endl;
        l1attn_cuda_forward_kernelX<scalar_t><<<numBlocks, threadsPerBlock>>>(
            q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            scale, bs, n_ctx, n_heads, width);
    }));
    
    return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_forward16(
        torch::Tensor q,
        torch::Tensor k) {
  
    int bs = q.sizes()[0]; 
    int n_heads = q.sizes()[1];
    int n_ctx = q.sizes()[2]; 
    int width = q.sizes()[3];
    
    auto options = torch::TensorOptions()
        .dtype(q.dtype())
        .device(q.device())
        .requires_grad(q.requires_grad()); //better way to do this?
    
    auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
    
    const dim3 numBlocks(n_ctx/BLKSIZ, n_ctx/BLKSIZ, bs*n_heads); // x, y, z
    const dim3 threadsPerBlock(BLKSIZ, BLKSIZ, 1);
    
    double scale = -1.0 / sqrt(width); 
        
    AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel16", ([&] {
        if (width == 16) {
            // std::cout << "CUDA: Executing forward kernel for width 16" << std::endl;
            l1attn_cuda_forward_kernel16_w16<scalar_t><<<numBlocks, threadsPerBlock>>>(
                q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                scale, bs, n_ctx, n_heads, width);
        } else if (width == 32) {
            // std::cout << "CUDA: Executing forward kernel for width 32" << std::endl;
            l1attn_cuda_forward_kernel16_w32<scalar_t><<<numBlocks, threadsPerBlock>>>(
                q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                scale, bs, n_ctx, n_heads, width);
        } else if (width == 64) {
            // std::cout << "CUDA: Executing forward kernel for width 64" << std::endl;
            l1attn_cuda_forward_kernel16_w64<scalar_t><<<numBlocks, threadsPerBlock>>>(
                q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                scale, bs, n_ctx, n_heads, width);
        } else {
            throw std::runtime_error("Unsupported width for l1attn_cuda_forward16");
        }
    }));
    
    return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_backward(
        torch::Tensor d_attnq,
        torch::Tensor d_attnk,
        torch::Tensor q,
        torch::Tensor k) 
{
    int bs = q.sizes()[0]; // permuted in python driver!!!
    int width = q.sizes()[1];
    int n_heads = q.sizes()[2]; 
    int n_ctx = q.sizes()[3]; 
    
    double scale = -1.0 / sqrt(width);
    
    auto options = torch::TensorOptions()
        .dtype(q.dtype())
        .device(q.device())
        .requires_grad(q.requires_grad());
    
    auto d_q = torch::zeros({bs, n_ctx, n_heads, width}, options);
    auto d_k = torch::zeros({bs, n_ctx, n_heads, width}, options);
    
    const dim3 numBlocks(n_heads*n_ctx, width, bs); // x, y, z
    const dim3 threadsPerBlock(32, 1, 1);
    
    AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel", ([&] {
        // std::cout << "CUDA: Executing general backward kernel" << std::endl;
        l1attn_cuda_backward_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
            d_attnq.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            d_attnk.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            scale, bs, n_ctx, n_heads, width);
    }));
    
    return {d_q, d_k};
}

std::vector<torch::Tensor> l1attn_cuda_backward16(
        torch::Tensor d_attn,
        torch::Tensor q,
        torch::Tensor k) 
{
    int bs = q.sizes()[0]; 
    int n_heads = q.sizes()[1]; 
    int n_ctx = q.sizes()[2]; 
    int width = q.sizes()[3];
    
    double scale = -1.0 / sqrt(width);
    
    auto options = torch::TensorOptions()
        .dtype(q.dtype())
        .device(q.device())
        .requires_grad(q.requires_grad());
    
    auto d_q = torch::zeros({bs, n_heads, n_ctx, width}, options);
    auto d_k = torch::zeros({bs, n_heads, n_ctx, width}, options);
    
    const dim3 numBlocks(n_ctx/BLKSIZ, n_ctx/BLKSIZ, n_heads*bs); // x, y, z
    const dim3 threadsPerBlock(BLKSIZ, BLKSIZ, 1); 
    
    AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel16", ([&] {
        if (width == 16) {
            // std::cout << "CUDA: Executing backward kernel for width 16" << std::endl;
            l1attn_cuda_backward_kernel16_w16<scalar_t><<<numBlocks, threadsPerBlock>>>(
                d_attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                scale, bs, n_ctx, n_heads, width);
        } else if (width == 32) {
            // std::cout << "CUDA: Executing backward kernel for width 32" << std::endl;
            l1attn_cuda_backward_kernel16_w32<scalar_t><<<numBlocks, threadsPerBlock>>>(
                d_attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                scale, bs, n_ctx, n_heads, width);
        } else if (width == 64) {
            // std::cout << "CUDA: Executing backward kernel for width 64" << std::endl;
            l1attn_cuda_backward_kernel16_w64<scalar_t><<<numBlocks, threadsPerBlock>>>(
                d_attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                scale, bs, n_ctx, n_heads, width);
        } else {
            throw std::runtime_error("Unsupported width for l1attn_cuda_backward16");
        }
    }));
    
    // bhtw -> bthw -- really need to change everything in the lib! 
    d_q = d_q.transpose_(1,2).contiguous();
    d_k = d_k.transpose_(1,2).contiguous(); 
    
    return {d_q, d_k}; // reduce along the zsize dim
>>>>>>> 11dc1eb (new files)
}
