#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__device__  __forceinline__ scalar_t sign(scalar_t x)
{ 
	scalar_t t = x < 0 ? -1 : 0;
	return x > 0 ? 1 : t;
}

template <typename scalar_t>
__device__  __forceinline__ void fastAtomicAdd2(
	torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> out, 
	int i0, int i1, int i2, int i3, scalar_t v)
{
	// convenience wrapper function around
	// fastAtomicAdd for 4-D tensors. 
	int index = i0*out.stride(0) + i1*out.stride(1) + i2*out.stride(2) + i3*out.stride(3);
	at::native::fastAtomicAdd(out.data(), index, 1, v, true); 
}

template <typename scalar_t>
__global__ void l1attn_cuda_forward_kernelX(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		attn,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width)
{
	__shared__ scalar_t acc[8][32];
	
	int tix = threadIdx.x; // [0 .. 31]. 
	int h = threadIdx.y; // n_heads
	// tix operates within across the width dimension (reduction dim) 
	int t = blockIdx.x; 
	int s = blockIdx.y; 
	int b = blockIdx.z; 
	
	/*
		// we can permute the order of the output indexing here to improve
		// memory gather coherency.  
		// but, because each warp can only write one mem loc, 
		// it's still a gather operation.
		// empirical notes: permuting the indexing order did not change speed! 
		int j = indx; 
		int h = j % n_heads; 
		j /= n_heads; 
		int t = j % n_ctx; 
		j /= n_ctx; 
		int s = j % n_ctx; 
		j /= n_ctx; 
		int b = j; */
	
	int width32 = (width + 31) / 32; 
	scalar_t f = 0.0; 
	for(int w = 0; w < width32; w++) { 
		int o = w*32+tix; 
		if(o < width)
			f += abs(q[b][t][h][o] - k[b][s][h][o]); 
	}
	acc[h][tix] = f * scale; 
	if(tix < 16) { 
		acc[h][tix] += acc[h][tix + 16];
		__syncthreads(); // why is this needed ??? 
		acc[h][tix] += acc[h][tix + 8 ];
		__syncthreads(); // threads in a warp should be synchronous.
		acc[h][tix] += acc[h][tix + 4 ];
		__syncthreads(); // experiment: it's totally needed! 
		acc[h][tix] += acc[h][tix + 2 ];
		__syncthreads();
		acc[h][tix] += acc[h][tix + 1 ];
		__syncthreads();
		if(tix == 0){
			attn[b][s][t][h] = acc[h][tix]; 
		}
	}
}

// template <typename scalar_t>
// __global__ void l1attn_cuda_backward_kernel_old(
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_attn,
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		q,
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		k,
// 		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_q,
// 		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_k,
// 		const scalar_t scale, 
// 		const int bs, const int n_ctx, const int n_heads, const int width ) 
// {
// 	// reduction (across s and t) has to be done within a thread warp: 
// 	// can't have different warps write the same memory. 
// 	// they will interfere / not give the correct answer!
// 	
// 	int indx = threadIdx.x + blockIdx.x * blockDim.x; // 1D
// 	
// 	if(indx < bs*n_ctx*n_ctx*n_heads){
// 		// again, output indexing b/c thread blocks can't overlap writes.
// 		// see note in forward kernel.
// 		int j = indx; 
// 		int h = j % n_heads; 
// 		j /= n_heads; 
// 		int s = j % n_ctx; 
// 		j /= n_ctx; 
// 		int t = j % n_ctx; 
// 		j /= n_ctx; 
// 		int b = j % bs; 
// 		
// 		scalar_t d_a = d_attn[b][s][t][h]; 
// 		for(int w = 0; w < width; w++){
// 			scalar_t ws = q[b][t][h][w] - k[b][s][h][w];
// 			ws = sign(ws) * scale; 
// 			// atomicAdd((scalar_t*)&(d_q[b][t][h][w]), ws * d_a);
// 			// atomicAdd((scalar_t*)&(d_k[b][s][h][w]), -1*ws * d_a);
// 			fastAtomicAdd2(d_q, b,t,h,w, ws * d_a);
// 			fastAtomicAdd2(d_k, b,s,h,w, -1*ws * d_a);
// 		}
// 	}
// } 

template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attnq,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attnk,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_q,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_k,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width ) 
{
	__shared__ scalar_t acc_dq[8][32];
	__shared__ scalar_t acc_dk[8][32];
	
	// int tix = threadIdx.x; // [0 .. 31]. 
	// int tiy = threadIdx.y; // [0 .. 7]
	// // tix operates within across the width dimension (reduction dim) 
	// int indx = threadIdx.y + blockIdx.x * 8; // is this right?
	
	int tix = threadIdx.x; // [0 .. 31].
	int h = threadIdx.y; // n_heads
	int u = blockIdx.x; // u is t for q, s for k.
	int w = blockIdx.y; 
	int b = blockIdx.z; 
	
	// if(indx < bs*n_ctx*n_heads*width){
	// 	// again, output indexing b/c thread blocks can't overlap writes.
	// 	// see note in forward kernel.
	// 	int j = indx; 
	// 	int u = j % n_ctx; 
	// 	j /= n_ctx; 
	// 	int h = j % n_heads; 
	// 	j /= n_heads; 
	// 	int w = j % width; 
	// 	j /= width; 
	// 	int b = j % bs; 
		
	int ctx32 = (n_ctx + 31) / 32; 
	scalar_t dq = 0.0; 
	scalar_t dk = 0.0; 
	
	scalar_t qq = q[b][w][h][u]; 
	for(int o = 0; o < ctx32; o++) { 
		int s = o*32+tix; 
		if(s < n_ctx){ 
			// all this would work better if n_ctx were a multiple of 32. 
			scalar_t ws = qq - k[b][w][h][s];
			ws = sign(ws) * scale; 
			scalar_t d_a = d_attnq[b][u][h][s]; 
			dq += ws * d_a; 
		}
	}
	
	scalar_t kk = k[b][w][h][u]; 
	for(int o = 0; o < ctx32; o++) { 
		int t = o*32+tix; 
		if(t < n_ctx){
			scalar_t ws = q[b][w][h][t] - kk;
			ws = sign(ws) * scale; 
			scalar_t d_a = d_attnk[b][u][h][t]; 
			dk -= ws * d_a; 
		}
	}
	
	acc_dq[h][tix] = dq;
	acc_dk[h][tix] = dk;
	if(tix < 16) { 
		acc_dq[h][tix] += acc_dq[h][tix + 16];
		acc_dk[h][tix] += acc_dk[h][tix + 16];
		__syncthreads(); 
		acc_dq[h][tix] += acc_dq[h][tix + 8 ];
		acc_dk[h][tix] += acc_dk[h][tix + 8 ];
		__syncthreads(); 
		acc_dq[h][tix] += acc_dq[h][tix + 4 ];
		acc_dk[h][tix] += acc_dk[h][tix + 4 ];
		__syncthreads();
		acc_dq[h][tix] += acc_dq[h][tix + 2 ];
		acc_dk[h][tix] += acc_dk[h][tix + 2 ];
		__syncthreads();
		acc_dq[h][tix] += acc_dq[h][tix + 1 ];
		acc_dk[h][tix] += acc_dk[h][tix + 1 ];
		__syncthreads();
		if(tix == 0){
			d_q[b][u][h][w] = acc_dq[h][tix];
			d_k[b][u][h][w] = acc_dk[h][tix]; 
		}
	}
}

std::vector<torch::Tensor> l1attn_cuda_forward(
		torch::Tensor q,
		torch::Tensor k) {
  
	int bs = q.sizes()[0]; 
	int n_ctx = q.sizes()[1]; 
	int n_heads = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad()); //better way to do this? 
	
	auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
	
	const dim3 numBlocks(n_ctx, n_ctx, bs); // x, y, z
	// const int n_elements = bs * n_heads * n_ctx * n_ctx; 
	// int n_blocks = (n_elements + 7) / 8;
	// int n_blocks = n_elements;
	// int n_threads = 32;
	const dim3 threadsPerBlock(32, n_heads, 1);
	
	double scale = -1.0 / sqrt(width); 
		
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel", ([&] {
		l1attn_cuda_forward_kernelX<scalar_t><<<numBlocks, threadsPerBlock>>>(
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_backward(
		torch::Tensor d_attnq,
		torch::Tensor d_attnk,
		torch::Tensor q,
		torch::Tensor k) 
{
	int bs = q.sizes()[0]; // permuted in python driver!!!
	int width = q.sizes()[1];
	int n_heads = q.sizes()[2]; 
	int n_ctx = q.sizes()[3]; 
	
	double scale = -1.0 / sqrt(width);
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad());
	
	auto d_q = torch::zeros({bs, n_ctx, n_heads, width}, options);
	auto d_k = torch::zeros({bs, n_ctx, n_heads, width}, options);
	
	// const dim3 dimBlocks(32, 8); // x, y, z
	const dim3 numBlocks(n_ctx, width, bs); // x, y, z
	const dim3 threadsPerBlock(32, n_heads, 1);
	// const int n_elements = bs * n_heads * n_ctx * width; 
	// int n_blocks = n_elements;
	// int n_threads = 32;
	
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel", ([&] {
		l1attn_cuda_backward_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
			d_attnq.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_attnk.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {d_q, d_k};
}
